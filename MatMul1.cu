#include "hip/hip_runtime.h"
#include<stdio.h>
//Cuda headers
#include<hip/hip_runtime.h>
#include"helper_timer.h"
//macros
#define BLOCK_WIDTH 32

//global variables
int *hostA=NULL;
int *hostB=NULL;
int *hostC=NULL;
int *gold=NULL;

int  *deviceA=NULL;
int *deviceB=NULL;
int *deviceC=NULL;
float timeOnCPU=0.0f;
float timeOnGPU=0.0f;

//cuda kernel function
__global__ void matMulGPU(int *A,int *B,int *C,int numARows,int numAColumns,int numBColumns,int numCColumns)
{

	//variable declarations
	int row=blockIdx.y * blockDim.y+threadIdx.y;
	int column=blockIdx.x*blockDim.x+threadIdx.x;
	//code
	if((row < numARows) && (column < numBColumns))
	{
	int value=0.0;
	for(int k=0;k<numAColumns;k++)
	{
		int a=A[row*numAColumns+k];
		int b=B[k*numBColumns+column];
		value+=a*b;
	}
	C[row*numCColumns+column]=value;
	}
}
int main(int argc,char *argv[])
{
	//function declarations
	void InitA(int *data,int,int);
	void InitB(int *data,int,int);
	void matMulCPU(int*,int*,int*,int,int,int,int);
	void cleanup(void);
	//variable declarations
	int numARows=BLOCK_WIDTH;
	int numAColumns=BLOCK_WIDTH;
	int numBRows=BLOCK_WIDTH;
	int numBColumns=BLOCK_WIDTH;
	int numCRows=numARows;
	int numCColumns=numBColumns;

	int numGoldRows=numARows;
	int numGoldColumns=numBColumns;

	int sizeA=numARows * numAColumns * sizeof(int);
	int sizeB=numBRows * numBColumns * sizeof(int);
	int sizeC=numCRows * numCColumns * sizeof(int);
	int sizeGold=numGoldRows * numGoldColumns*sizeof(int);
	
	hipError_t result=hipSuccess;
	//code
	//host memory allocation
	hostA=(int *)malloc(sizeA);
	if(hostA==NULL)
	{
		printf("Host Memory Allocation is failed for hostA matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	hostB =(int *)malloc(sizeB);
	if(hostB==NULL)
	{
		printf("Host Memory Allocation is failed for hostB matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	hostC =(int *)malloc(sizeC);
	if(hostC==NULL)
	{
		printf("Host Memory Allocation is failed for hostC matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	gold=(int*)malloc(sizeGold);
	if(gold==NULL)
	{
		printf("Host Memory allocation is failed for gold matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);

	}
	//printing matrix dimensions and sizes
	printf("The Dimensions of Matrix 'hostA' Are: %d X %d\n",numARows,numAColumns);
	printf("The Dimensions of Matrix 'hostB' Are: %d X %d\n",numBRows,numBColumns);
	printf("The Dimensions of Matrix 'hostC' Are: %d X %d\n",numCRows,numCColumns);
	printf("size of Matrix gold= %d\n",sizeGold);
	//fill source matrices
	InitA(hostA,numARows,numAColumns);
	InitB(hostB,numBRows,numBColumns);
	//device memory allocation
	result=hipMalloc((void**)&deviceA,sizeA);
	if(result!=hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceA matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	result=hipMalloc((void**)&deviceB,sizeB);
	if(result!=hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceB matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	result=hipMalloc((void**)&deviceC,sizeC);
	if(result!=hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceC matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	//Copy data from host matrices into device matrices
	result=hipMemcpy(deviceA,hostA,sizeA,hipMemcpyHostToDevice);
	if(result!=hipSuccess)
	{
		printf("Host to Device Data Copy is failed for deviceA matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	//CUDA kernel configuration
	dim3 dimGrid=dim3(ceil((int)numBColumns/(int)BLOCK_WIDTH),ceil((int)numARows/(int)BLOCK_WIDTH),1);
	dim3 dimBlock=dim3(BLOCK_WIDTH,BLOCK_WIDTH,1);

	//CUDA kernel for matrix multiplication
	StopWatchInterface* timer =NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	matMulGPU<<<dimGrid,dimBlock>>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBColumns,numCColumns);
	sdkStopTimer(&timer);
	timeOnGPU =sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer=NULL;
	//copy data from device matrix into host matrix
	result=hipMemcpy(hostC,deviceC,sizeC,hipMemcpyDeviceToHost);
	if(result!=hipSuccess)
	{
		printf("Device to host data copy is failed for hostC matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	//matrix multiplication on host
	matMulCPU(hostA, hostB, gold, numARows, numAColumns, numBColumns, numCColumns);
	//comparison
	int breakValue=-1;
	bool bAccuracy=true;
	for(int i=0;i<numCRows*numCColumns;i++)
	{
		int val1=gold[i];
		int val2=hostC[i];
		if(val1!=val2)
		{
			bAccuracy=false;
			breakValue=i;
			break;
		}
	}
	char str[128];
	if(bAccuracy==false)
		sprintf(str,"Comparison of CPU and GPU Matrix Multiplication is not accurate at array index %d",breakValue);
		else
		sprintf(str,"Comparison of CPU and GPU Matrix Multiplication is accurate");
		printf("Time taken for matrix Multiplication on CPU =%.6f\n",timeOnCPU);
		printf("TIme taken for matrix multiplication on GPU=%.6f\n",timeOnGPU);
		printf("%s\n",str);

		//cleanup
		cleanup();
		return (0);
}

void InitA(int *data,int row,int col)
{
	int num=1;
	//code
	for(int i=0;i<row;i++)
	{
		for(int j=0;j<col;j++)
		{
			*(data+i*col+j)=num;
			num++;
		}
	}
}

void InitB(int *data,int row,int col)
{
	int num=BLOCK_WIDTH;
	//code
	for(int i=0;i<row;i++)
	{
		for(int j=0;j<col;j++)
		{
			*(data+i*col+j)=num;
			num--;
		}
	}
}
void matMulCPU(int* A,int* B,int* C,int numARows,int numAColumns,int numBColumns,int numCColumns)
{
	//code
	StopWatchInterface* timer=NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	for(int i=0;i<numARows;i++)
	{
		for(int j=0;j<numBColumns;j++)
		{
			int value=0.0f;
			for(int k=0;k<numAColumns;k++)
			{
				int a=A[i*numAColumns+k];
				int b=B[k*numBColumns+j];
				value +=a*b;
			}
			C[i*numCColumns +j]=value;
		}
	}
	sdkStopTimer(&timer);
	timeOnCPU=sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer=NULL;
}
void cleanup(void)
{
//code
if(deviceC)
{
	hipFree(deviceC);
	deviceC=NULL;
}
if(deviceB)
{
	hipFree(deviceB);
	deviceB=NULL;
}
if(deviceA)
{
	hipFree(deviceA);
	deviceA=NULL;
}
if(gold)
{
	free(gold);
	gold=NULL;

}
if(hostC)
{
	free(hostC);
	hostC=NULL;
}
if(hostB)
{
	free(hostB);
	hostB=NULL;
}
if(hostA)
{
	free(hostA);
	hostA=NULL;
}
}
